#include <stdio.h>
#include <iostream>
#include "HaarCascade.h"
#include "opencv2/objdetect/objdetect.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"

using namespace std;

int main(int argc, char** argv){
	unsigned int height = 512;
	unsigned int width = 1024;

	//GENERATE HAAR CLASSIFIER FROM OPENCV FILE
	cv::CascadeClassifier classifierOpenCV;
	classifierOpenCV.load("./data/haarcascade_frontalface_default.xml");
	void * test = cvLoad("./data/haarcascade_frontalface_default.xml");
	CvHaarClassifierCascade * haarCascade = (CvHaarClassifierCascade *) cvLoad("./data/haarcascade_frontalface_default.xml");
	HaarCascade hc_cpu = HaarCascade(haarCascade, height, width);


	//TEMPORARILY USE OPENCV INTEGRAL IMAGE
	//TODO: CHANGE THIS SECTION
	cv::Mat image = cv::Mat(height, width, CV_8UC1);
	for (int i = 0; i < 1024 * 512; i++) {
		image.data[i] = 1;
	}

	cv::Mat sum = cv::Mat(height+1, width+1, CV_32FC1);
	cv::Mat sqsum = cv::Mat(height + 1, width + 1, CV_32FC1);

	cv::integral(image, sum, sqsum);

	//LOAD DATA TO GPU
	HaarCascade hc_gpu = HaarCascade(hc_cpu);
	hc_cpu.copyToGPU(&hc_gpu);
	float *gpu_sum;
	double * gpu_sqsum;
	CHECK(hipMalloc(&gpu_sum, sizeof(float)*(height + 1)*(width + 1)));
	CHECK(hipMalloc(&gpu_sqsum, sizeof(double)*(height + 1)*(width + 1)));
	CHECK(hipMemcpy(gpu_sum, sum.data, sizeof(float)*(height + 1)*(width + 1), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(gpu_sqsum, sqsum.data, sizeof(double)*(height + 1)*(width + 1), hipMemcpyHostToDevice));
	

	//ALLOCATE SPACES FOR DETECTED FACES
	Rect * faces, faces_gpu;
	faces = (Rect *)malloc(width*height * sizeof(Rect));
	memset(faces, 0, width*height * sizeof(Rect));

	CHECK(hipMalloc(&faces_gpu, width*height * sizeof(Rect)));
	CHECK(hipMemcpy(faces_gpu, faces, width*height * sizeof(Rect), hipMemcpyHostToDevice));

	//Determine image scales
	//TODO: CHANGE THIS SO THAT IT DOESN'T LOOK IDENTICAL
	double curr_scale = 1.0f;
	float scale_mult = 1.2f;
	std::vector<double> scale;

	while (curr_scale*hc_cpu.obj_width < width - 10 &&
		curr_scale*hc_cpu.obj_height < height - 10) {
		scale.push_back(curr_scale);
		curr_scale *= scale_mult;
	}

	for (int i = 0; i < scale.size(); i++) {
		//COPY SCALED FILTERS TO DEVICE
		hc_cpu.scaleFilters(scale[i]);
		CHECK(hipMemcpy(hc_gpu.scaled_filters, hc_cpu.scaled_filters, hc_cpu.num_filters * sizeof(Filter), hipMemcpyHostToDevice));

		hc_gpu.

	}

}
