#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include "cpuViolaJones.hpp"
#include "paths.hpp"
#include "gpuViolaJones.cuh"
#include "load_images.hpp"

using namespace std;


int main(int argc, char** argv)
{
	cv::Mat image;
	image = cv::imread(FACE_PATH, 1);
	cv::String face_cascade_path = CASCADE_PATH;
	if (!image.data)
	{
		printf("No image data \n");
		return -1;
	}

	cv::Mat gray_face;
	cv::cvtColor(image, gray_face, CV_BGR2GRAY);
	unsigned int height = 0;
	unsigned int width = 0;
	height = gray_face.rows;
	width = gray_face.cols;
	unsigned int large = max(height, width);
	if (height > 1024 || width > 1024) {
		float large_scale = 1024.0 / large;
		cv::resize(gray_face, gray_face, Size(), large_scale, large_scale);
		height = gray_face.rows;
		width = gray_face.cols;
	}
	imshow("gray", gray_face);
	unsigned char * face;
	if (gray_face.isContinuous()) {
		face = gray_face.data;
		height = gray_face.rows;
		width = gray_face.cols;
	}
	else {
		fprintf(stderr, "Stop\n");
	}

	// Load test images
	printf("Loading image set\n");
	int *numImgs = new int;

	//TEST CODE
	unsigned int width1 = 100;
	unsigned int height1 = 100;
	unsigned char * input = new unsigned char[width1 * height1];
	for (int i = 0; i < width1 * height1; i++) {
		input[i] = i + 1;
	}
	unsigned int min_size = 24;
	float scale = 1.2;
	//END TEST CODE
	unsigned char * result = gpuViolaJones(face, width, height, 24, 1.2);

	Mat result_img = Mat(height, width, CV_8U, result);
	imshow("test", result_img);

	cpuViolaJones(image, face_cascade_path);
	waitKey(0);

	return 0;
}
